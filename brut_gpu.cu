#include <hip/hip_runtime.h>

#include <iostream>
#include <chrono>
#include <stdio.h>
#include <sstream>



#define arraySize 31 //35 max
//#define W 1741

using namespace std;

__constant__ float coefs[arraySize*2+1];
__global__ void single_thread(float *sh_sum_dev,long int *str_num_dev, float num_of_blocks, int rep,int threads_per_block,int max_blocks)
{
  float th_w_sum = 0;
   float th_v_sum = 0;
   float th_bin[arraySize];
   int n_of_it = rep;
  extern __shared__ float sh_array[];
  float* sh_maxs = (float*)sh_array;
  long int* indices = (long int*)&sh_maxs[threads_per_block];
  indices[threadIdx.x] = threadIdx.x;


long signed int num_to_bin = blockIdx.x * blockDim.x + threadIdx.x;
num_to_bin += max_blocks * n_of_it;
__syncthreads();
#pragma unroll
  for (uint i = 0; i < arraySize; i++)
    {
      th_bin[i] = ((num_to_bin) >> i) % 2;
      th_w_sum += th_bin[i] * coefs[i];
      th_v_sum += th_bin[i] * coefs[i+arraySize];
    }

sh_maxs[threadIdx.x] = (th_w_sum > coefs[arraySize*2]) ? 0:th_v_sum;

__syncthreads ();

  for (uint offset = blockDim.x >> 1; offset >= 1; offset >>= 1)
    {
      if (threadIdx.x < offset)
	{
	  if (sh_maxs[threadIdx.x] < sh_maxs[threadIdx.x + offset])
	    {
	      sh_maxs[threadIdx.x] = sh_maxs[threadIdx.x + offset];
	      indices[threadIdx.x] = indices[threadIdx.x + offset];
	    }
	}
      __syncthreads ();
    }
  // write result for this block to global mem
  if(threadIdx.x == 0){
  sh_sum_dev[blockIdx.x+max_blocks*rep] = sh_maxs[0];
  //str_num_dev[blockIdx.x+max_blocks*rep] = indices[0]+max_blocks*rep;
}
if(threadIdx.x == indices[0]){str_num_dev[blockIdx.x+max_blocks*rep] = num_to_bin;}

}

__global__ void
reduction_max (float *s, long int *str_num_dev,int threads_per_block)
{
  int ID = blockIdx.x * blockDim.x + threadIdx.x;
  extern __shared__ float sdata[];
  sdata[threadIdx.x] = s[ID];
  sdata[threadIdx.x + threads_per_block] = str_num_dev[ID];

  __syncthreads ();
  // do reduction in shared mem
  for (uint s = blockDim.x >>1; s > 0; s >>= 1)
    {
      if (threadIdx.x < s)
	{
	  if (sdata[threadIdx.x] < sdata[threadIdx.x + s])
	    {
	      sdata[threadIdx.x] = sdata[threadIdx.x + s];
	      sdata[threadIdx.x + threads_per_block] =
		sdata[threadIdx.x + threads_per_block + s];
	    }
	}
      __syncthreads ();
    }
  // write result for this block to global mem
  if (threadIdx.x == 0)
    {
			//if(sdata[0]>s[0]){//}&&(blockIdx.x>0)){
      s[blockIdx.x] = sdata[0];
      str_num_dev[blockIdx.x] = sdata[threads_per_block];
		}

    //}
}

__global__ void
which_string (int a, int *view_dev)
{
  view_dev[threadIdx.x] = (a >> threadIdx.x) % 2;
}


int main(){
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  int threads_per_block = deviceProp.maxThreadsDim[0];
  int max_blocks = deviceProp.maxGridSize[0]/2 + 1;
  long int strSize_b = pow (2, arraySize);
  int num_of_blocks = strSize_b / threads_per_block;
  float *Sum = new float[1];	// = { 0 };
  float *sh_sum_dev;

  string line;
  float v;
  float* dev_coefs = new float[arraySize*2+1];
  while(getline(cin,line)){
    istringstream iss(line);
    int q = 0;
    while(iss>>v){
      dev_coefs[q] = v;
      q++;
}


  //float *values_dev;
  long int *str_num_dev;
  long int *str_num =  new long int[1];
  float N_of_rep;
  N_of_rep = num_of_blocks/max_blocks>0?num_of_blocks/max_blocks:1;
  int sing_blocks = num_of_blocks/N_of_rep>0?num_of_blocks/N_of_rep:1;

//for(int i = 0;i<arraySize*2;i++){dev_coefs[i] = 2;}

  std::chrono::time_point<std::chrono:: high_resolution_clock> start, end;

      start = std::chrono::high_resolution_clock::now();

  hipMalloc ((void **) &sh_sum_dev,  num_of_blocks * sizeof (float));
  hipMalloc ((void **) &str_num_dev, num_of_blocks * sizeof (long));
  hipMemcpyToSymbol (HIP_SYMBOL(coefs), dev_coefs, (2*arraySize + 1) * sizeof (float));



        for(int i = 0;i<N_of_rep;i++){
          //cout<<i;
  single_thread <<< sing_blocks, threads_per_block,threads_per_block*3*sizeof(int) >>> (sh_sum_dev, str_num_dev, num_of_blocks,i,threads_per_block,max_blocks);
             }

int k = num_of_blocks/threads_per_block;
while(k>=1){
//cout<<k<<" ";

               if(k>=threads_per_block){
                reduction_max <<<k, threads_per_block,threads_per_block*3*sizeof(int)>>> (sh_sum_dev, str_num_dev,threads_per_block);
                 k/=threads_per_block;}
               else break;
             }
if(k>1){
reduction_max <<<1,k,k*2*sizeof(int)>>> (sh_sum_dev, str_num_dev,k);
}

  hipMemcpy (Sum, sh_sum_dev, sizeof (float), hipMemcpyDeviceToHost);
  hipMemcpy (str_num, str_num_dev, sizeof (float), hipMemcpyDeviceToHost);

  end = std::chrono:: high_resolution_clock::now();

      int elapsed_seconds = std::chrono::duration_cast<std::chrono::microseconds>
                               (end-start).count();
      std::time_t end_time = std::chrono::system_clock::to_time_t(end);

      std::cout<< "Время выполнения: " << elapsed_seconds << "microseconds\n";

  cout << "Acheived maximal sum = " << Sum[0] << "\n";
  cout << "String number " << int(str_num[0]) << "\n";

  int *view = new int[arraySize];
  int *view_dev;
  hipMalloc ((void **) &view_dev, arraySize * sizeof (int));
  which_string <<< 1, arraySize >>> (str_num[0], view_dev);
  hipMemcpy (view, view_dev, arraySize * sizeof (int),
	      hipMemcpyDeviceToHost);
  for (int i = 0; i < arraySize; i++)
    {
      cout << view[i] << " ";
    } cout << "\n";
  //check
  float checksum = 0;
  for (int i = 0; i < arraySize; i++)
    {
      checksum += dev_coefs[i+arraySize] * view[i];
    }
  cout << "Validation sum = " << checksum << "\n";
  checksum = 0;
  for (int i = 0; i < arraySize; i++)
    {
      checksum += dev_coefs[i] * view[i];
    } cout << "Weight = " << checksum << "\n";

  hipFree (sh_sum_dev);
  hipFree (str_num_dev);
  hipFree (coefs);
  hipFree (view_dev);
}
  return 0;
}
