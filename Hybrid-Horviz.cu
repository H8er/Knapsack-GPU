
#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>
#include <sstream>

#define arraySize 31 // 35 max
#define def_div 10   // 5<=X<=15
//#define W 100
//#define threads_per_block 32
//#define max_blocks 32

using namespace std;

__constant__ float coefs[arraySize * 2 + 1];
__global__ void hybrid(float *sh_sum_dev, long int *str_num_dev,
                       float num_of_blocks, int *bdevX, int *global_mem_bin,
                       int threads_per_block) {
  float th_w_sum = 0;
  float th_v_sum = 0;
  int th_bin[arraySize];
  int best_bin[arraySize];
  extern __shared__ float sh_array[];
  float *sh_maxs = (float *)sh_array;
  int *indices = (int *)&sh_maxs[threads_per_block];
  int reached = 0;
  indices[threadIdx.x] = blockIdx.x * blockDim.x + threadIdx.x;
  __syncthreads();

  long signed int num_to_bin = blockIdx.x * blockDim.x + threadIdx.x;
// num_to_bin += max_blocks * n_of_it;
#pragma unroll
  for (uint i = 0; i < def_div; i++) {
    th_bin[i] = ((num_to_bin) >> i) % 2;
    th_w_sum += th_bin[i] * coefs[i];
    th_v_sum += th_bin[i] * coefs[i + arraySize];
    best_bin[i] = th_bin[i];
  }
#pragma unroll
  for (uint i = def_div; i < arraySize; i++) {
    th_bin[i] = -1;
  }
  int Capacity = coefs[arraySize * 2] - th_w_sum;
  sh_maxs[threadIdx.x] = (th_w_sum > coefs[arraySize * 2]) ? 0 : th_v_sum;
  __syncthreads();

  // H_S
  int h = def_div;
  long int ns = 0;
  bool forward;

  while (h - def_div != -1) {
    ns++;
    forward = true;
    if (th_bin[h] == -1) {
      th_bin[h] = 1;
    } else {
      if (th_bin[h] == 1) {
        th_bin[h] = 0;
      } else {
        if (th_bin[h] == 0) {
          th_bin[h] = -1;
          h--;
          forward = false;
        }
      }
    }
    if (h == arraySize - 1) {
      int cw = 0;
      int cp = 0;
#pragma unroll
      for (int i = def_div; i < arraySize; i++) {
        cp += coefs[i + arraySize] * th_bin[i];
        cw += coefs[i] * th_bin[i];
      }
      if ((cw <= Capacity) && (cp > reached)) {
        reached = cp;
#pragma unroll
        for (int i = def_div; i < arraySize; i++) {
          best_bin[i] = th_bin[i];
        }
      }
    } else {
      int cw = 0;
      for (int i = def_div; i < arraySize; i++) {
        cw += coefs[i] * th_bin[i];
      }
      if (cw > Capacity)
        forward = false;
      cw = 0;
      float cp = 0;
      int nw = 0;
      int np = 0;
#pragma unroll
      for (int i = def_div; i < arraySize; i++) {
        np = th_bin[i] != -1 ? th_bin[i] * coefs[i + arraySize]
                             : coefs[i + arraySize];
        nw = th_bin[i] != -1 ? th_bin[i] * coefs[i] : coefs[i];
        if (cw + nw <= Capacity) {
          cw += nw;
          cp += np;
        } else {
          cp += np * (Capacity - cw) / nw;
          break;
        }
      }
      int b = cp;
      if (b <= reached) {
        forward = false;
      }
    }
    if (forward) {
      if (h < arraySize - 1) {
        h++;
      }
    }
  }

  sh_maxs[threadIdx.x] += reached;

  __syncthreads();
  // reduction on block
  for (uint offset = blockDim.x >> 1; offset >= 1; offset >>= 1) {
    if (threadIdx.x < offset) {
      if (sh_maxs[threadIdx.x] < sh_maxs[threadIdx.x + offset]) {
        sh_maxs[threadIdx.x] = sh_maxs[threadIdx.x + offset];
        indices[threadIdx.x] = indices[threadIdx.x + offset];
      }
    }
    __syncthreads();
  }
  // write result for this block to global mem
  if (threadIdx.x == 0) {
    sh_sum_dev[blockIdx.x] = sh_maxs[0];
    str_num_dev[blockIdx.x] = indices[0];
  }
  if (blockIdx.x * blockDim.x + threadIdx.x == indices[0]) {
#pragma unroll
    for (int i = 0; i < arraySize; i++) {
      global_mem_bin[blockIdx.x * arraySize + i] = best_bin[i];
    }
  }
  __syncthreads();
}

__global__ void hybrid_reduction(float *s, long int *str_num_dev,
                                 int *global_mem_bin, int threads_per_block) {
  int ID = blockIdx.x * blockDim.x + threadIdx.x;
  extern __shared__ int sh_hy_data[];
  sh_hy_data[threadIdx.x] = s[ID];
  sh_hy_data[threadIdx.x + threads_per_block] = str_num_dev[ID];

  __syncthreads();
  // do reduction in shared mem
  for (uint s = blockDim.x >> 1; s > 0; s >>= 1) {
    if (threadIdx.x < s) {
      if (sh_hy_data[threadIdx.x] < sh_hy_data[threadIdx.x + s]) {
        sh_hy_data[threadIdx.x] = sh_hy_data[threadIdx.x + s];
        sh_hy_data[threadIdx.x + threads_per_block] =
            sh_hy_data[threadIdx.x + threads_per_block + s];
      }
    }
    __syncthreads();
  }
  // write result for this block to global mem
  if (threadIdx.x == 0) {
    // if(sh_hy_data[0]>s[0]){//}&&(blockIdx.x>0)){
    s[blockIdx.x] = sh_hy_data[0];
    str_num_dev[blockIdx.x] = sh_hy_data[threads_per_block];

#pragma unroll
    for (int i = 0; i < arraySize; i++) {
      global_mem_bin[i] =
          global_mem_bin[(sh_hy_data[threads_per_block] / threads_per_block) *
                             arraySize +
                         i];
    }
  }
}

__global__ void which_string(long int a, int *view_dev) {
  view_dev[threadIdx.x] = (a >> threadIdx.x) % 2;
}

void quickSortR(float *a, float *b, long N) {
  // На входе - массив a[], a[N] - его последний элемент.

  long i = 0, j = N; // поставить указатели на исходные места
  float temp, p;

  p = a[N >> 1]; // центральный элемент

  // процедура разделения
  do {
    while (a[i] > p)
      i++;
    while (a[j] < p)
      j--;

    if (i <= j) {
      temp = a[i];
      a[i] = a[j];
      a[j] = temp;
      temp = b[i];
      b[i] = b[j];
      b[j] = temp;
      temp = b[i + arraySize];
      b[i + arraySize] = b[j + arraySize];
      b[j + arraySize] = temp;
      i++;
      j--;
    }
  } while (i <= j);

  // рекурсивные вызовы, если есть, что сортировать
  if (j > 0)
    quickSortR(a, b, j);
  if (N > i)
    quickSortR(a + i, b + i, N - i);
}

int main() {
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  int threads_per_block = deviceProp.warpSize;
  int max_blocks = pow(2, def_div) / threads_per_block;
  long int strSize_b = pow(2, arraySize);
  int num_of_blocks = strSize_b / threads_per_block;
  float *Sum = new float[32]; // = { 0 };
  float *sh_sum_dev;

  int iter = 0;

  string line;
  float v;
  float *dev_coefs = new float[arraySize * 2 + 1];
  while (getline(cin, line)) {
    istringstream iss(line);
    int q = 0;
    while (iss >> v) {
      dev_coefs[q] = v;
      q++;
    }
    cout<<"Iter = "<<iter<<"\n";iter++;
    for (int i = 0; i < arraySize * 2 + 1; i++) {
      cout << dev_coefs[i] << " ";
    }
    cout << "\n";
    // int W = int(dev_coefs[arraySize*2]);

    long int *str_num_dev;
    long int *str_num = new long int[1];

    float *additional_array = new float[arraySize];
    for (int i = 0; i < arraySize; i++) {
      additional_array[i] = dev_coefs[i + arraySize] / dev_coefs[i];
    }

    quickSortR(additional_array, dev_coefs, arraySize - 1);

    int *bdevX;
    hipMalloc((void **)&bdevX, arraySize * sizeof(int));
    int *global_mem_bin;
    hipMalloc((void **)&global_mem_bin, max_blocks * arraySize * sizeof(int));

    hipMalloc((void **)&sh_sum_dev, num_of_blocks * sizeof(float));
    hipMalloc((void **)&str_num_dev, num_of_blocks * sizeof(long));
    hipMemcpyToSymbol(HIP_SYMBOL(coefs), dev_coefs, (2 * arraySize + 1) * sizeof(float));

    std::chrono::time_point<std::chrono::high_resolution_clock> start, end;
    start = std::chrono::high_resolution_clock::now();



    hybrid<<<max_blocks, threads_per_block,
             threads_per_block * 3 * sizeof(int)>>>(
        sh_sum_dev, str_num_dev, num_of_blocks, bdevX, global_mem_bin,
        threads_per_block);

    hybrid_reduction<<<1, max_blocks, threads_per_block * 3 * sizeof(int)>>>(
        sh_sum_dev, str_num_dev, global_mem_bin, threads_per_block);

    int *suda = new int[arraySize];
    hipMemcpy(Sum, sh_sum_dev, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(str_num, str_num_dev, sizeof(long int), hipMemcpyDeviceToHost);
    hipMemcpy(suda, global_mem_bin, arraySize * sizeof(int),
               hipMemcpyDeviceToHost);

    end = std::chrono::high_resolution_clock::now();

    int elapsed_seconds =
        std::chrono::duration_cast<std::chrono::microseconds>(end - start)
            .count();
    std::time_t end_time = std::chrono::system_clock::to_time_t(end);

    std::cout << "Время выполнения: " << elapsed_seconds << "microseconds\n";
    cout << "Acheived maximal sum = " << Sum[0] << "\n";
    cout << str_num[0] << "\n";
    for (int i = 0; i < arraySize; i++) {
      cout << suda[i];
    }
    cout << "\n";

    // check
    int checksum = 0;
    for (int i = 0; i < arraySize; i++) {
      checksum += dev_coefs[i + arraySize] * suda[i];
    }
    cout << "Validation sum = " << checksum << "\n";
    checksum = 0;
    for (int i = 0; i < arraySize; i++) {
      checksum += dev_coefs[i] * suda[i];
    }
    cout << "Weight = " << checksum << "\n";

    hipFree(coefs);
    hipFree(sh_sum_dev);
    hipFree(str_num_dev);
    hipFree(bdevX);
    hipFree(global_mem_bin);
/*
    delete[] Sum;
    delete[] suda;
    delete[] str_num;
    delete[] dev_coefs;
    delete[] additional_array;
*/
    //cudaDeviceReset();
  }
  return 0;
}
