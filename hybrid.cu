
#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <fstream>

#define arraySize 31 //35 max
#define def_div 10  // 5<=X<=15
#define W 100
//#define threads_per_block 32
//#define max_blocks 32

using namespace std;

__constant__ float coefs[arraySize*2];
__global__ void hybrid(float *sh_sum_dev, long int *str_num_dev, float num_of_blocks, int* bdevX,int* global_mem_bin,int threads_per_block)
{
  float th_w_sum = 0;
   float th_v_sum = 0;
   int th_bin[arraySize];
   int best_bin[arraySize];
   extern __shared__ float sh_array[];
   float* sh_maxs = (float*)sh_array;
   int* indices = (int*)&sh_maxs[threads_per_block];
  int reached = 0;
  indices[threadIdx.x] = blockIdx.x * blockDim.x + threadIdx.x;
  __syncthreads();

long signed int num_to_bin = blockIdx.x * blockDim.x + threadIdx.x;
//num_to_bin += max_blocks * n_of_it;
#pragma unroll
  for (uint i = 0; i < def_div; i++)
    {
      th_bin[i] = ((num_to_bin) >> i) % 2;
      th_w_sum += th_bin[i] * coefs[i];
      th_v_sum += th_bin[i] * coefs[i+arraySize];
      best_bin[i] = th_bin[i];
    }
#pragma unroll
    for (uint i = def_div; i < arraySize; i++)
      {
        th_bin[i] = -1;
      }
int Capacity = W - th_w_sum;
sh_maxs[threadIdx.x] = (th_w_sum > W) ? 0:th_v_sum;
__syncthreads ();

//H_S
int h = def_div;
long int ns = 0;
bool forward;

while(h-def_div!=-1){
  ns++;
  forward = true;
  if(th_bin[h]==-1){
     th_bin[h]=1;
  }else{
  if(th_bin[h]==1){
     th_bin[h]=0;
  }else{
  if(th_bin[h]==0){
     th_bin[h]=-1;
    h--;
    forward=false;
  }
}
}
  if(h==arraySize-1){
    int cw = 0;
    int cp = 0;
    #pragma unroll
    for(int i = def_div;i<arraySize;i++){
      cp += coefs[i+arraySize] * th_bin[i];
      cw += coefs[i] * th_bin[i];
    }
    if((cw <= Capacity) &&(cp > reached)){
      reached = cp;
      #pragma unroll
      for(int i = def_div; i < arraySize; i++){
        best_bin[i] = th_bin[i];
      }
    }
  }
  else{
    int cw = 0;
    for(int i = def_div ; i < arraySize; i++){
      cw += coefs[i] * th_bin[i];
    }
    if (cw > Capacity) forward = false;
    cw = 0;
    float cp = 0;
    int nw = 0;
    int np = 0;
    #pragma unroll
    for(int i = def_div;i < arraySize;i++){
      np = th_bin[i]!=-1? th_bin[i] * coefs[i+arraySize]:coefs[i+arraySize];
      nw = th_bin[i]!=-1? th_bin[i] * coefs[i]: coefs[i];
      if(cw+nw <= Capacity){
        cw += nw;
        cp += np;
      }
      else{
        cp+=np*(Capacity-cw)/nw;
        break;
      }
    }
    int b = cp;
    if (b <= reached){
      forward = false;
    }
  }
  if(forward){if(h<arraySize-1){h++;}
              }
  }

sh_maxs[threadIdx.x] += reached;

__syncthreads();
//reduction on block
  for (uint offset = blockDim.x >> 1; offset >= 1; offset >>= 1)
    {
      if (threadIdx.x < offset)
	{
	  if (sh_maxs[threadIdx.x] < sh_maxs[threadIdx.x + offset])
	    {
	      sh_maxs[threadIdx.x] = sh_maxs[threadIdx.x + offset];
	      indices[threadIdx.x] = indices[threadIdx.x + offset];
	    }
	}
      __syncthreads ();
    }
  // write result for this block to global mem
  if(threadIdx.x == 0){
  sh_sum_dev[blockIdx.x] = sh_maxs[0];
  str_num_dev[blockIdx.x] = indices[0];
  }
  if(blockIdx.x*blockDim.x+threadIdx.x == indices[0]){
    #pragma unroll
    for(int i = 0; i < arraySize;i++){
      global_mem_bin[blockIdx.x*arraySize + i] = best_bin[i];
  }
  }
  __syncthreads();
}

__global__ void
hybrid_reduction (float *s, long int *str_num_dev,int* global_mem_bin,int threads_per_block)
{
  int ID = blockIdx.x * blockDim.x + threadIdx.x;
  extern __shared__ int sh_hy_data[];
  sh_hy_data[threadIdx.x] = s[ID];
  sh_hy_data[threadIdx.x + threads_per_block] = str_num_dev[ID];

  __syncthreads ();
  // do reduction in shared mem
  for (uint s = blockDim.x >>1; s > 0; s >>= 1)
    {
      if (threadIdx.x < s)
	{
	  if (sh_hy_data[threadIdx.x] < sh_hy_data[threadIdx.x + s])
	    {
	      sh_hy_data[threadIdx.x] = sh_hy_data[threadIdx.x + s];
	      sh_hy_data[threadIdx.x + threads_per_block] =
		sh_hy_data[threadIdx.x + threads_per_block + s];
	    }
	}
      __syncthreads ();
    }
  // write result for this block to global mem
  if (threadIdx.x == 0)
    {
			//if(sh_hy_data[0]>s[0]){//}&&(blockIdx.x>0)){
      s[blockIdx.x] = sh_hy_data[0];
      str_num_dev[blockIdx.x] = sh_hy_data[threads_per_block];

            #pragma unroll
            for(int i = 0; i < arraySize;i++){
             global_mem_bin[i] = global_mem_bin[(sh_hy_data[threads_per_block]/arraySize)*arraySize + i];

          }
		}

}


__global__ void
which_string (long int a, int *view_dev)
{
  view_dev[threadIdx.x] = (a>>threadIdx.x)%2;
}


void quickSortR(float* a,float* b, long N) {
// На входе - массив a[], a[N] - его последний элемент.

    long i = 0, j = N;      // поставить указатели на исходные места
    float temp, p;

    p = a[ N>>1 ];      // центральный элемент

    // процедура разделения
    do {
        while ( a[i] > p ) i++;
        while ( a[j] < p ) j--;

        if (i <= j) {
            temp = a[i]; a[i] = a[j]; a[j] = temp;
            temp = b[i]; b[i] = b[j]; b[j] = temp;
            temp = b[i+arraySize]; b[i+arraySize] = b[j+arraySize]; b[j+arraySize] = temp;
            i++; j--;
        }
    } while ( i<=j );

    // рекурсивные вызовы, если есть, что сортировать
    if ( j > 0 ) quickSortR(a,b, j);
    if ( N > i ) quickSortR(a+i,b+i, N-i);
}


    int main(){
      hipDeviceProp_t deviceProp;
      hipGetDeviceProperties(&deviceProp, 0);
      int threads_per_block = deviceProp.warpSize;
      int max_blocks = pow(2,def_div)/threads_per_block;
      long int strSize_b = pow (2, arraySize);
      int num_of_blocks = strSize_b / threads_per_block;
      float *Sum = new float[32];	// = { 0 };
      float *sh_sum_dev;
      //float weight[31] ={ 5, 10, 17, 19, 20, 23, 26, 30, 32, 38, 40, 44, 47, 50, 55, 56, 56, 60, 62, 66, 70, 75, 77, 80, 81, 90,93,96,101,107,115 };
      //float values[31] ={ 10, 13, 16, 22, 30, 25, 55, 90, 110, 115, 130, 120, 150, 170, 194, 199, 194, 199, 217, 230, 248, 250, 264, 271, 279, 286,293,299,305,313,321 };
      float dev_coefs[62] = {2,1,8,2,17,22,21,33,54,53,29,34,91,24,82,91,51,9,64,14,44,30,23,98,38,55,98,64,57,80,66,49,24,89,15,87,86,77,81,89,82,44,38,86,22,75,72,40,7,47,9,28,17,10,42,15,20,32,15,6,4,1};

      //float dev_coefs[60] = {5, 10, 17, 19, 20, 23, 26, 30, 32, 38, 40, 44, 47, 50, 55, 56, 56, 60, 62, 66, 70, 75, 77, 80, 81, 90,93,96,101,107, 10, 13, 16, 22, 30, 25, 55, 90, 110, 115, 130, 120, 150, 170, 194, 199, 194, 199, 217, 230, 248, 250, 264, 271, 279, 286,293,299,305,313 };
      //float dev_coefs[58] = {5, 10, 17, 19, 20, 23, 26, 30, 32, 38, 40, 44, 47, 50, 55, 56, 56, 60, 62, 66, 70, 75, 77, 80, 81, 90,93,96,101, 10, 13, 16, 22, 30, 25, 55, 90, 110, 115, 130, 120, 150, 170, 194, 199, 194, 199, 217, 230, 248, 250, 264, 271, 279, 286,293,299,305 };

      //float *values_dev;
      long int *str_num_dev;
      long int *str_num = new long int[1];

      cout<<"sing param = "<<max_blocks<<" _ "<< threads_per_block<<"\n";
      cout<<"red param "<<1<<"  ,  "<<max_blocks<<"\n";

      float* additional_array = new float[arraySize];
      for(int i = 0; i < arraySize;i++){
      additional_array[i] = dev_coefs[i+arraySize]/dev_coefs[i];
      }

      quickSortR(additional_array,dev_coefs,arraySize-1);

	float t1,t2;
	float acceleration = 0;

      //for(int i = 0;i<arraySize*2;i++){dev_coefs[i] = 2;}

      std::chrono::time_point<std::chrono:: high_resolution_clock> start, end;
          start = std::chrono::high_resolution_clock::now();

      int* bdevX;
      hipMalloc ((void **) &bdevX, arraySize * sizeof (int));
      int* global_mem_bin;
      hipMalloc ((void **) &global_mem_bin, max_blocks*arraySize * sizeof (int));





      hipMalloc ((void **) &sh_sum_dev,  num_of_blocks * sizeof (float));
      hipMalloc ((void **) &str_num_dev, num_of_blocks * sizeof (float));
      hipMemcpyToSymbol (HIP_SYMBOL(coefs), dev_coefs, 2*arraySize * sizeof (float));




       hybrid <<< max_blocks, threads_per_block ,threads_per_block*2*sizeof(int)>>> (sh_sum_dev, str_num_dev, num_of_blocks,bdevX,global_mem_bin,threads_per_block);



hybrid_reduction<<<1,max_blocks,threads_per_block*2*sizeof(int)>>>(sh_sum_dev,str_num_dev,global_mem_bin,threads_per_block);
int* suda = new int[arraySize];
      hipMemcpy (Sum, sh_sum_dev, sizeof (int), hipMemcpyDeviceToHost);
      hipMemcpy (str_num, str_num_dev, sizeof (long int), hipMemcpyDeviceToHost);
      hipMemcpy (suda, global_mem_bin, arraySize*sizeof (int), hipMemcpyDeviceToHost);


      end = std::chrono:: high_resolution_clock::now();

          int elapsed_seconds = std::chrono::duration_cast<std::chrono::microseconds>
                                   (end-start).count();
          std::time_t end_time = std::chrono::system_clock::to_time_t(end);

          std::cout<< "Время выполнения: " << elapsed_seconds << "microseconds\n";
	t1 = elapsed_seconds;
      cout << "Acheived maximal sum = " << Sum[0] << "\n";

        for (int i = 0; i < arraySize; i++)
          {
            cout << suda[i];
          } cout << "\n";


        //check
        int checksum = 0;
        for (int i = 0; i < arraySize; i++)
          {
            checksum += dev_coefs[i+arraySize] * suda[i];
          }
        cout << "Validation sum = " << checksum << "\n";
        checksum = 0;
        for (int i = 0; i < arraySize; i++)
          {
            checksum += dev_coefs[i] * suda[i];
          } cout << "Weight = " << checksum << "\n";
         // ofstream fout;
         // fout.open("data_uncorr_hybrid.txt",ios_base::app);

         // fout<<"GPU\n"<<Sum[0]<<"\n"<<elapsed_seconds<<"\n\n";



        hipFree(coefs);
        hipFree (sh_sum_dev);
        hipFree (str_num_dev);
        hipFree(bdevX);
        hipFree(global_mem_bin);

        delete [] Sum;
        delete [] str_num;


        cout<<"Проверка. CPU version:\n";
        start = std::chrono::high_resolution_clock::now();
        int *X = new int[arraySize];
        int *bestX = new int[arraySize];
        for(int i = 0; i < arraySize; i++){
          X[i] = -1;
          bestX[i] = 0;
        }
        int curr_sum = 0;
        int reached_max = 0;
        float *cpu_bin = new float[arraySize];

        for(int i = 0; i < arraySize;i++){
        additional_array[i] = dev_coefs[i+arraySize]/dev_coefs[i];
        }
        quickSortR(additional_array,dev_coefs,arraySize-1);

        int h = 0;
        int k = h;//def_div;
        long int ns = 0;
        bool forward;
        while(h-k!=-1){
          ns++;
          forward = true;
          if(X[h]==-1){
            X[h]=1;
          }else{
          if(X[h]==1){
            X[h]=0;
          }else{
          if(X[h]==0){
            X[h]=-1;
            h--;
            forward=false;
          }
        }
        }
          if(h==arraySize-1){
            int cw = 0;
            int cp = 0;
            for(int i = k;i<arraySize;i++){
              cp += dev_coefs[i+arraySize]*X[i];
              cw += dev_coefs[i]*X[i];
            }
            if((cw <= W) &&(cp > reached_max)){
              reached_max = cp;
              for(int i = k; i < arraySize; i++){
                bestX[i] = X[i];
              }
            }
          }
          else{
            int cw = 0;
            for(int i = k ; i < arraySize; i++){
              cw += dev_coefs[i]*X[i];
            }
            if (cw > W) forward = false;
            cw = 0;
            float cp = 0;
            int nw = 0;
            int np = 0;
            for(int i = k;i<arraySize;i++){
              np = X[i]!=-1? X[i] * dev_coefs[i+arraySize]:dev_coefs[i+arraySize];
              nw = X[i]!=-1? X[i] * dev_coefs[i]: dev_coefs[i];
              if(cw+nw <= W){
                cw += nw;
                cp += np;
              }
              else{
                cp+=np*(W-cw)/nw;
                break;
              }
            }
            int b = cp;
            if (b <= reached_max){
              forward = false;
            }
          }
          if(forward){if(h<arraySize-1){h++;}}
          }


          end = std::chrono:: high_resolution_clock::now();

              elapsed_seconds = std::chrono::duration_cast<std::chrono::microseconds>
                                       (end-start).count();
               end_time = std::chrono::system_clock::to_time_t(end);
		t2 = elapsed_seconds;
          std::cout<< "Время выполнения: " << elapsed_seconds << "microseconds\n";

        cout<<"MAX = "<<reached_max<<"\n";
        for(int m = 0 ; m < arraySize;m++){
        cout<<bestX[m];
        curr_sum += bestX[m]*dev_coefs[m+arraySize];
        }cout<<"\nЧисло итераций = "<<ns<<"\n";


       // fout<<"CPU\n"<<reached_max<<"\n"<<elapsed_seconds<<"\n\n";
      //  fout.close();
acceleration = t2/t1;
cout<<"Acceleration = "<<acceleration<<"\n";
delete [] suda;
delete [] additional_array;

hipFree (sh_sum_dev);
hipFree (str_num_dev);
hipFree (coefs);

return 0;
}
